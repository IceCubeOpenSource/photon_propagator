#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "util.cu"
#include "gamma.cu"
#include "rotate.cu"

// Given a set of photons, detector configuration, and an ice model
// propagate the photons and generate hits.
__global__ void propagate(configuration* config,			  
			  hiprandState_t* rng_state,
			  ices* ice_model,
			  DOM* oms,
			  photon* photons,
			  unsigned int photon_buffer_size, // number of photons to propagate
			  hit* hits,
			  unsigned int hit_buffer_size){			  

  const unsigned thread_index = blockIdx.x + threadIdx.x * gridDim.x;
  /* Copy state to local memory for efficiency */
  hiprandState_t local_rng_state = rng_state[thread_index];
  hiprandState_t* state_ptr = &local_rng_state;

  // TODO: At some point figure out how to properly fill
  //       the hit buffer in a thread safe way.
  //       We can't just map photons to hits.
  //       We're paying for photon space anyway.
  //       Can't we just figure out which ones hit
  //       an OM and mark them? ...I think we pay for
  //       off-loading every single photon, and it's expensive.
  
  //unsigned& hit_index = e.hidx;
  
  // maybe split into multiple kernels?
  float sca; // scattering length
  int old;
  float TOT=0;
  for(unsigned int i=thread_index;
      i<photon_buffer_size;
      TOT==0 && (i=atomicAdd(&hit_index, gridDim.x)) ){
    // this 'increment' statement is a little evil
    //   TOT==0 && (i=atomicAdd(&hit_index, gridDim.x)
    // it increments hit_index by the gridDim.x only when TOT is 0.
    // remember the hit_index is a reference to the global hidx
    // in the global 'dats' (i.e. context) struct.

    int om=-1;
    float3 n={0,0,0};   // photon direction
    float4 r={0,0,0,0}; // photon position
    ices* w;
    if(TOT==0){ // initialize photon
      photon f=photons[i];
      r=f.r;
      n=f.n;
      w = &ice_model[f.q]; // this allows us to get the ice layer...erp.
      om = -1;
      TOT = -logf(hiprand_uniform(state_ptr));
      sca=0;
    }

    // if there is any time TOT is non-zero
    // sca is not initialized and this is undefined behavior.
    // it will randomly and almost always evaluate to false.
    if(sca==0){ // get distance for overburden
      float SCA = -logf(hiprand_uniform(state_ptr));
      old=om;
      float tot;
      float z=r.z;
      float nr=1.f;

      int i=__float2int_rn((z-e.hmin)*e.rdh);
      if(i<0) i=0; else if(i>=e.size) i=e.size-1;
      float h=e.hmin+i*e.dh; // middle of the layer
      h=n.z<0?h-e.hdh:h+e.hdh;

      float ais=(n.z*SCA-(h-z)*w->ice_properties[i].sca)*e.rdh;
      float aia=(n.z*TOT-(h-z)*w->ice_properties[i].abs)*e.rdh;

      int j=i;
      if(n.z<0) for(; j>0 && ais<0 && aia<0; h-=e.dh, ais+=w->ice_properties[j].sca, aia+=w->ice_properties[j].abs) --j;
      else for(; j<e.size-1 && ais>0 && aia>0; h+=e.dh, ais-=w->ice_properties[j].sca, aia-=w->ice_properties[j].abs) ++j;

      if(i==j || fabsf(n.z)<XXX) sca=SCA/w->ice_properties[j].sca, tot=TOT/w->ice_properties[j].abs;
      else sca=(ais*e.dh/w->ice_properties[j].sca+h-z)/n.z, tot=(aia*e.dh/w->ice_properties[j].abs+h-z)/n.z;

      // get overburden for distance
      if(tot<sca){
	sca=tot;
	TOT=0;
      }else{
	TOT=nr*(tot-sca)*w->ice_properties[j].abs;
      }
    }

    om=-1;
    float del=sca;      // W.
    { // sphere         // T.
      float & sca = del;// F. ???
      float2 ri, rf, pi, pf;

      ri.x=r.x;
      rf.x=r.x+sca*n.x;
      
      ri.y=r.y;
      rf.y=r.y+sca*n.y;

      ctr(e, ri, pi);
      ctr(e, rf, pf);

      ri.x=min(pi.x, pf.x)-e.rx; rf.x=max(pi.x, pf.x)+e.rx;
      ri.y=min(pi.y, pf.y)-e.rx; rf.y=max(pi.y, pf.y)+e.rx;

      int2 xl, xh;

      xl.x=min(max(__float2int_rn((ri.x-e.cl[0])*e.crst[0]), 0), e.cn[0]);
      xh.x=max(min(__float2int_rn((rf.x-e.cl[0])*e.crst[0]), e.cn[0]-1), -1);

      xl.y=min(max(__float2int_rn((ri.y-e.cl[1])*e.crst[1]), 0), e.cn[1]);
      xh.y=max(min(__float2int_rn((rf.y-e.cl[1])*e.crst[1]), e.cn[1]-1), -1);

      // i'm thinking we just calculated cell boundaries
      
      for(int i=xl.x, j=xl.y;
	  i<=xh.x && j<=xh.y;
	  ++j<=xh.y?:(j=xl.y, i++))
	for(unsigned char k=e.is[i][j]; k!=0x80; ){
	  unsigned char m=e.ls[k];
	  line & s = e.sc[m&0x7f];
	  k=m & 0x80 ? 0x80:k+1;

	  float b=0, c=0, dr;
	  dr=s.x-r.x;
	  b+=n.x*dr; c+=dr*dr;
	  dr=s.y-r.y;
	  b+=n.y*dr; c+=dr*dr;

	  float np=1-n.z*n.z;
	  float D=b*b-(c-s.r*s.r)*np;
	  if(D>=0){
	    D=sqrtf(D);
	    float h1=b-D, h2=b+D;
	    if(h2>=0 && h1<=sca*np){
	      if(np>XXX){
		h1/=np;
		h2/=np;
		if(h1<0) h1=0;
		if(h2>sca) h2=sca;
	      }else {
		h1=0;
		h2=sca;
	      }
	      
	      h1=r.z+n.z*h1, h2=r.z+n.z*h2;
	      float zl, zh;
	      if(n.z>0){
		zl=h1;
		zh=h2;
	      }else{
		zl=h2;
		zh=h1;
	      }

	      int omin=0, omax=s.max;
	      int n1=s.n-omin+min(omax+1, max(omin, __float2int_ru(omin-(zh-s.dl-s.h)*s.d)));
	      int n2=s.n-omin+max(omin-1, min(omax, __float2int_rd(omin-(zl-s.dh-s.h)*s.d)));
	      
	      for(int l=n1; l<=n2; l++) if(l!=old){
		  if(l==-1) 
		    continue;
		  const DOM & dom=oms[l];
		  float b=0, c=0, dr;
		  dr=dom.r[0]-r.x;
		  b+=n.x*dr; c+=dr*dr;
		  dr=dom.r[1]-r.y;
		  b+=n.y*dr; c+=dr*dr;
		  dr=dom.r[2]-r.z;
		  b+=n.z*dr; c+=dr*dr;
		  float D=b*b-c+e.R2;
		  if(D>=0){
		    float sqd=sqrtf(D);
		    float h=b-sqd*e.zR;
		    if(h>0 && h<=del) om=l, del=h;
		  }
		}
	    }
	  }
	}
    }
    
    sca-=del;
    { // advance
      r.x+=del*n.x;
      r.y+=del*n.y;
      r.z+=del*n.z;
      r.w+=del*w->ocm;
    }

    if(!isfinite(TOT) || !isfinite(sca)){
      TOT=0;
      om=-1;
    }

    float xi=hiprand_uniform(state_ptr);
    if(om!=-1){
      // this means we have a hit!
      // create a hit object and add
      // it to the hit buffer.
      
      hit h;
      h.i=om;
      h.t=r.w;
      h.z=w->wvl;

      float sum;
      {
	float& x = n.z;
	float y=1;
	sum=e.s[0];
	for(int i=1; i<ANUM; i++){
	  y*=x;
	  sum+=e.s[i]*y;
	}
      }

      if(e.mas*xi<sum){
	unsigned int j = atomicAdd(&ed->hidx, 1);
	if(j<e.hit_buffer_size)
	  hits[j]=h;
	  // and if it's greater?
	  // we just gloss over that?
      }

      if(e.zR==1) TOT=0; else old=om;
    }
    else if(TOT<XXX) TOT=0;
    else{
      float &sf=e.sf, &g=e.g, &g2=e.g2, &gr=e.gr;

      if(xi>sf){
	xi=(1-xi)/(1-sf);
	xi=2*xi-1;
	if(g!=0){
	  float ga=(1-g2)/(1+g*xi);
	  xi=(1+g2-ga*ga)/(2*g);
	}
      }
      else{
	xi/=sf;
	xi=2*powf(xi, gr)-1;
      }

      if(xi>1) xi=1; else if(xi<-1) xi=-1;

      float si=sqrtf(1-xi*xi);
      rotate(xi, si, n, state_ptr);

    } // what does this close?
  } // what does this close?
  // This is the end of the photon propagation loop
  //  for(unsigned int i=thread_index;
  //      i<photon_buffer_size;
  //      TOT==0 && (i=atomicAdd(&hit_index, gridDim.x)) ){

  /* Copy state back to global memory */
  rng_state[thread_index] = local_rng_state;
  
  __syncthreads();
  __threadfence();
}
